#include "hip/hip_runtime.h"
// This code contains NVIDIA Confidential Information and is disclosed 
// under the Mutual Non-Disclosure Agreement. 
// 
// Notice 
// ALL NVIDIA DESIGN SPECIFICATIONS AND CODE ("MATERIALS") ARE PROVIDED "AS IS" NVIDIA MAKES 
// NO REPRESENTATIONS, WARRANTIES, EXPRESSED, IMPLIED, STATUTORY, OR OTHERWISE WITH RESPECT TO 
// THE MATERIALS, AND EXPRESSLY DISCLAIMS ANY IMPLIED WARRANTIES OF NONINFRINGEMENT, 
// MERCHANTABILITY, OR FITNESS FOR A PARTICULAR PURPOSE. 
// 
// NVIDIA Corporation assumes no responsibility for the consequences of use of such 
// information or for any infringement of patents or other rights of third parties that may 
// result from its use. No license is granted by implication or otherwise under any patent 
// or patent rights of NVIDIA Corporation. No third party distribution is allowed unless 
// expressly authorized by NVIDIA.  Details are subject to change without notice. 
// This code supersedes and replaces all information previously supplied. 
// NVIDIA Corporation products are not authorized for use as critical 
// components in life support devices or systems without express written approval of 
// NVIDIA Corporation. 
// 
// Copyright � 2008- 2013 NVIDIA Corporation. All rights reserved.
//
// NVIDIA Corporation and its licensors retain all intellectual property and proprietary
// rights in and to this software and related documentation and any modifications thereto.
// Any use, reproduction, disclosure or distribution of this software and related
// documentation without an express license agreement from NVIDIA Corporation is
// strictly prohibited.
//

#include "Shared_Globals.h"
#include <cassert>

template <typename T> T sqr(const T& t) { return t*t; }
__device__ float2 operator+(const float2& v0, const float2& v1) { return make_float2(v0.x + v1.x, v0.y + v1.y); }
__device__ float2 operator-(const float2& v0, const float2& v1) { return make_float2(v0.x - v1.x, v0.y - v1.y); }
__device__ float2 operator*(const float2& v, const float& s) { return make_float2(v.x * s, v.y * s); }
__device__ float2 make_float2(const float& s) { return make_float2(s, s); }

struct Constants
{
	float2* m_Gauss;
	float2* m_H0;
	float2* m_Ht;
	float4* m_Dt;
	float* m_Omega;

	int m_resolution;
	int m_resolution_plus_one;
	int m_half_resolution;
	int m_half_resolution_plus_one;
	int m_half_of_resolution_squared;
	int m_resolution_plus_one_squared_minus_one;
	int m_32_minus_log2_resolution;

	float m_window_in;
	float m_window_out;

	float m_frequency_scale;
	float m_linear_scale;
	float m_wind_scale;
	float m_root_scale;
	float m_power_scale;
	float2 m_wind_dir;
	float m_choppy_scale;
};

static __constant__ Constants gConstants[MAX_NUM_CASCADES];

extern "C" 
hipError_t cuda_GetConstantsSize(size_t* size)
{
	return hipGetSymbolSize(size, HIP_SYMBOL(gConstants));
}

extern "C" 
hipError_t cuda_GetConstantsAddress(void** ptr)
{
	return hipGetSymbolAddress(ptr, HIP_SYMBOL(gConstants));
}

extern "C" 
hipError_t cuda_SetConstants (void* dst,
						float2* Gauss, 
						float2* H0, 
						float2* Ht, 
						float4* Dt, 
						float* Omega,
						int resolution,
						float fft_period,
						float window_in,
						float window_out,
						float2 wind_dir,
						float wind_speed,
						float wind_dependency,
						float wave_amplitude,
						float small_wave_fraction,
						float choppy_scale,
						hipStream_t cu_stream)
{
	const float twoPi = 6.28318530718f;
	const float gravity = 9.810f;
	const float sqrtHalf = 0.707106781186f;
	const float euler = 2.71828182846f;

	float fftNorm = powf(float(resolution), -0.25f);
	float philNorm = euler / fft_period;
	float gravityScale = sqr(gravity / sqr(wind_speed));

	static Constants constants;
	constants.m_Gauss = Gauss;
	constants.m_H0 = H0;
	constants.m_Ht = Ht;
	constants.m_Dt = Dt;
	constants.m_Omega = Omega;
	constants.m_resolution = resolution;
	constants.m_resolution_plus_one = resolution+1;
	constants.m_half_resolution = resolution/2;
	constants.m_half_resolution_plus_one = resolution/2+1;
	constants.m_half_of_resolution_squared = sqr(resolution)/2;
	constants.m_resolution_plus_one_squared_minus_one = sqr(resolution+1)-1;
	for(int i = 0; (1 << i) <= resolution; ++i)
		constants.m_32_minus_log2_resolution = 32 - i;
	constants.m_window_in = window_in;
	constants.m_window_out = window_out;
	constants.m_wind_dir = wind_dir;
	constants.m_frequency_scale = twoPi / fft_period;
	constants.m_linear_scale = fftNorm * philNorm * sqrtHalf * wave_amplitude;
	constants.m_wind_scale = -sqrtf(1 - wind_dependency);
	constants.m_root_scale = -0.5f * gravityScale;
	constants.m_power_scale = -0.5f / gravityScale * sqr(small_wave_fraction);
	constants.m_choppy_scale = choppy_scale;

	return hipMemcpyAsync(dst, &constants, 
		sizeof(constants), hipMemcpyHostToDevice, cu_stream);
}

template <int N>
__global__ void kernel_ComputeH0()
{
	float2* __restrict__ h0_output = gConstants[N].m_H0;
	const float2* __restrict__ gauss_input = gConstants[N].m_Gauss;

	int columnIdx = blockIdx.x * blockDim.x + threadIdx.x;
	int rowIdx = blockIdx.y * blockDim.y + threadIdx.y;

	int nx = columnIdx - gConstants[N].m_half_resolution;
	int ny = rowIdx - gConstants[N].m_half_resolution;
	float nr = sqrtf(nx*nx + ny*ny);

	float amplitude = 0.0f;
	if((nx || ny) && nr >= gConstants[N].m_window_in && nr < gConstants[N].m_window_out)
	{
		float2 k = make_float2(nx, ny) * gConstants[N].m_frequency_scale;

		float kSqr = k.x * k.x + k.y * k.y;
		float kCos = k.x * gConstants[N].m_wind_dir.x + k.y * gConstants[N].m_wind_dir.y;

		float scale = gConstants[N].m_linear_scale * kCos * rsqrtf(kSqr * kSqr * kSqr);

		if (kCos < 0)
			scale *= gConstants[N].m_wind_scale;

		amplitude = scale * expf(gConstants[N].m_power_scale * kSqr + fdividef(gConstants[N].m_root_scale, kSqr));
	}

	int index = rowIdx * gConstants[N].m_resolution_plus_one + columnIdx;
	float2 h0 = gauss_input[index - rowIdx] * amplitude;
	h0_output[index] = h0;

	// mirror first row/column, CPU and CUDA paths don't do that
	// however, we need to initialize the N+1'th row/column to zero 
	if(!rowIdx || !columnIdx)
		h0_output[gConstants[N].m_resolution_plus_one_squared_minus_one - index] = make_float2(0); //h0;
}

extern "C" 
hipError_t cuda_ComputeH0(int resolution, int constantsIndex, hipStream_t cu_stream)
{
	dim3 block = dim3(8, 8); // block dimensions are fixed to be 64 threads
    dim3 grid = dim3(resolution / block.x, resolution / block.y);
	assert(grid.x * block.x == unsigned(resolution) && grid.y * block.y == unsigned(resolution));

	switch(constantsIndex)
	{
		case 0: kernel_ComputeH0<0><<<grid, block, 0, cu_stream>>>(); break;
		case 1: kernel_ComputeH0<1><<<grid, block, 0, cu_stream>>>(); break;
		case 2: kernel_ComputeH0<2><<<grid, block, 0, cu_stream>>>(); break;
		case 3: kernel_ComputeH0<3><<<grid, block, 0, cu_stream>>>(); break;
	}
	return hipPeekAtLastError();
}

extern __shared__ float2 gData[];

template <int N>
__device__ void fft(float2 (&u)[2], float2 (&v)[2], float2 (&w)[2])
{
	float2 u0 = u[0] + u[1], u1 = u[0] - u[1];
	float2 v0 = v[0] + v[1], v1 = v[0] - v[1];
	float2 w0 = w[0] + w[1], w1 = w[0] - w[1];

	int stride = 1;
	float scale = 3.14159265359f; // Pi

	#pragma unroll
	while(stride < 32)
	{
		bool flag = threadIdx.x & stride;

		float2 tu = flag ? u0 : u1;
		float2 tv = flag ? v0 : v1;
		float2 tw = flag ? w0 : w1;
#if __CUDA_ARCH__ >= 300
		tu.x = __shfl_xor(tu.x, stride);
		tu.y = __shfl_xor(tu.y, stride);
		tv.x = __shfl_xor(tv.x, stride);
		tv.y = __shfl_xor(tv.y, stride);
		tw.x = __shfl_xor(tw.x, stride);
		tw.y = __shfl_xor(tw.y, stride);
#else
		float2* pDst = gData + threadIdx.x;
		pDst[0]                               = tu;
		pDst[gConstants[N].m_half_resolution] = tv;
		pDst[gConstants[N].m_resolution]      = tw;
		__threadfence_block();
		float2* pSrc = gData + (threadIdx.x ^ stride);
		tu = pSrc[0];
		tv = pSrc[gConstants[N].m_half_resolution];
		tw = pSrc[gConstants[N].m_resolution];
#endif
		(flag ? u0 : u1) = tu;
		(flag ? v0 : v1) = tv;
		(flag ? w0 : w1) = tw;

		stride <<= 1;
		scale *= 0.5f;

		float sin, cos;
		int j = threadIdx.x & (stride-1);
		sincosf(j * scale, &sin, &cos);

		float2 du = make_float2(
			cos * u1.x - sin * u1.y, 
			sin * u1.x + cos * u1.y);
		float2 dv = make_float2(
			cos * v1.x - sin * v1.y, 
			sin * v1.x + cos * v1.y);
		float2 dw = make_float2(
			cos * w1.x - sin * w1.y, 
			sin * w1.x + cos * w1.y);

		u1 = u0 - du;
		u0 = u0 + du;
		v1 = v0 - dv;
		v0 = v0 + dv;
		w1 = w0 - dw;
		w0 = w0 + dw;
	}

	int i = threadIdx.x;
	while(stride < gConstants[N].m_half_resolution)
	{
		bool flag = threadIdx.x & stride;

		float2* pDst = gData + i;
		stride <<= 1;
		scale *= 0.5f;
		i = threadIdx.x ^ (stride - 32);
		float2* pSrc = gData + i;

		if(flag)
		{
			pDst[0]                               = u0;
			pDst[gConstants[N].m_half_resolution] = v0;
			pDst[gConstants[N].m_resolution]      = w0;
			__syncthreads();
			u0 = pSrc[0];
			v0 = pSrc[gConstants[N].m_half_resolution];
			w0 = pSrc[gConstants[N].m_resolution];
		}
		else 
		{
			pDst[0]                               = u1;
			pDst[gConstants[N].m_half_resolution] = v1;
			pDst[gConstants[N].m_resolution]      = w1;
			__syncthreads();
			u1 = pSrc[0];
			v1 = pSrc[gConstants[N].m_half_resolution];
			w1 = pSrc[gConstants[N].m_resolution];
		}

		float sin, cos;
		int j = threadIdx.x & (stride-1);
		sincosf(j * scale, &sin, &cos);

		float2 du = make_float2(
			cos * u1.x - sin * u1.y, 
			sin * u1.x + cos * u1.y);
		float2 dv = make_float2(
			cos * v1.x - sin * v1.y, 
			sin * v1.x + cos * v1.y);
		float2 dw = make_float2(
			cos * w1.x - sin * w1.y, 
			sin * w1.x + cos * w1.y);

		u1 = u0 - du;
		u0 = u0 + du;
		v1 = v0 - dv;
		v0 = v0 + dv;
		w1 = w0 - dw;
		w0 = w0 + dw;
	}

	u[0] = u0;
	u[1] = u1;
	v[0] = v0;
	v[1] = v1;
	w[0] = w0;
	w[1] = w1;
}

// update Ht, Dt_x, Dt_y from H0 and Omega, fourier transform per row (one CTA per row)
template <int N>
__launch_bounds__(MAX_FFT_RESOLUTION/2)
__global__ void kernel_ComputeRows(double timeOverTwoPi)
{
	float2* __restrict__ ht_output = gConstants[N].m_Ht;
	float4* __restrict__ dt_output = gConstants[N].m_Dt;
	const float2* __restrict__ h0_input = gConstants[N].m_H0;
	const float* __restrict__ omega_input = gConstants[N].m_Omega;

	int columnIdx = threadIdx.x * 2;
	int rowIdx = blockIdx.x;

	int reverseColumnIdx = __brev(columnIdx) >> gConstants[N].m_32_minus_log2_resolution;

	int nx = reverseColumnIdx - gConstants[N].m_half_resolution;
	int ny = reverseColumnIdx;
	int nz = rowIdx - gConstants[N].m_half_resolution;

	float2 h0i[2], h0j[2];
	double omega[2];

	int h0_index = rowIdx * gConstants[N].m_resolution_plus_one + reverseColumnIdx;
	int h0_jndex = h0_index + gConstants[N].m_half_resolution;
	int omega_index = rowIdx * gConstants[N].m_half_resolution_plus_one;
	int omega_jndex = omega_index + gConstants[N].m_half_resolution;

	h0i[0] = h0_input[h0_index];
	h0j[0] = h0_input[gConstants[N].m_resolution_plus_one_squared_minus_one - h0_index]; 
	omega[0] = omega_input[omega_index + reverseColumnIdx] * timeOverTwoPi;

	h0i[1] = h0_input[h0_jndex];
	h0j[1] = h0_input[gConstants[N].m_resolution_plus_one_squared_minus_one - h0_jndex]; 
	omega[1] = omega_input[omega_jndex - reverseColumnIdx] * timeOverTwoPi;

	float sinOmega[2], cosOmega[2];
	const float twoPi = 6.283185307179586476925286766559f;
	sincosf(float(omega[0] - floor(omega[0])) * twoPi, sinOmega + 0, cosOmega + 0);
	sincosf(float(omega[1] - floor(omega[1])) * twoPi, sinOmega + 1, cosOmega + 1);

	// H(0) -> H(t)
	float2 ht[2];
	ht[0].x = (h0i[0].x + h0j[0].x) * cosOmega[0] - (h0i[0].y + h0j[0].y) * sinOmega[0];
	ht[1].x = (h0i[1].x + h0j[1].x) * cosOmega[1] - (h0i[1].y + h0j[1].y) * sinOmega[1];
	ht[0].y = (h0i[0].x - h0j[0].x) * sinOmega[0] + (h0i[0].y - h0j[0].y) * cosOmega[0];
	ht[1].y = (h0i[1].x - h0j[1].x) * sinOmega[1] + (h0i[1].y - h0j[1].y) * cosOmega[1];

	float nrx = nx || nz ? rsqrtf(nx*nx + nz*nz) : 0;
	float nry = ny || nz ? rsqrtf(ny*ny + nz*nz) : 0;

	float2 dt0 = make_float2(-ht[0].y, ht[0].x) * nrx;
	float2 dt1 = make_float2(-ht[1].y, ht[1].x) * nry;

	float2 dx[2] = { dt0 * nx, dt1 * ny };
	float2 dy[2] = { dt0 * nz, dt1 * nz };

	fft<N>(ht, dx, dy);

	int index = rowIdx * gConstants[N].m_resolution + threadIdx.x;

	ht_output[index] = ht[0];
	ht_output[index+gConstants[N].m_half_resolution] = ht[1];

	dt_output[index] = make_float4(dx[0].x, dx[0].y, dy[0].x, dy[0].y);
	dt_output[index+gConstants[N].m_half_resolution] = make_float4(dx[1].x, dx[1].y, dy[1].x, dy[1].y);
}

extern "C" 
hipError_t cuda_ComputeRows(int resolution, double time, int constantsIndex, hipStream_t cu_stream)
{
	dim3 block = dim3(resolution/2);
    dim3 grid = dim3(resolution/2+1);
	int sharedMemory = 3 * sizeof(float) * resolution;

	const double oneOverTwoPi = 0.15915494309189533576888376337251;
	time *= oneOverTwoPi;

	switch(constantsIndex)
	{
		case 0: kernel_ComputeRows<0><<<grid, block, sharedMemory, cu_stream>>>(time); break;
		case 1: kernel_ComputeRows<1><<<grid, block, sharedMemory, cu_stream>>>(time); break;
		case 2: kernel_ComputeRows<2><<<grid, block, sharedMemory, cu_stream>>>(time); break;
		case 3: kernel_ComputeRows<3><<<grid, block, sharedMemory, cu_stream>>>(time); break;
	}
	return hipPeekAtLastError();
}

template <int N>
__device__ void computeColumns (float4 (&displacement_output)[2])
{
	const float2* __restrict__ ht_input = gConstants[N].m_Ht;
	const float4* __restrict__ dt_input = gConstants[N].m_Dt;

	int rowIdx = threadIdx.x * 2;
	int columnIdx = blockIdx.x;

	int reverseRowIdx = __brev(rowIdx) >> gConstants[N].m_32_minus_log2_resolution;

	int index = reverseRowIdx * gConstants[N].m_resolution + columnIdx;
	int jndex = (gConstants[N].m_half_resolution - reverseRowIdx) * gConstants[N].m_resolution + columnIdx;

	float2 ht[2];
	ht[0] = ht_input[index];
	ht[1] = ht_input[jndex];
	ht[1].y = -ht[1].y;

	float4 dti = dt_input[index];
	float4 dtj = dt_input[jndex];

	float2 dx[2] = { make_float2(dti.x, dti.y), make_float2(dtj.x, -dtj.y) };
	float2 dy[2] = { make_float2(dti.z, dti.w), make_float2(dtj.z, -dtj.w) };

	fft<N>(ht, dx, dy);

	float sgn = (threadIdx.x + columnIdx) & 0x1 ? -1.0f : +1.0f;
	float scale = gConstants[N].m_choppy_scale * sgn;

	displacement_output[0] = make_float4(dx[0].x * scale, dy[0].x * scale, ht[0].x * sgn, 0);
	displacement_output[1] = make_float4(dx[1].x * scale, dy[1].x * scale, ht[1].x * sgn, 0);
}

// do fourier transform per row of Ht, Dt_x, Dt_y, write displacement texture (one CTA per column)
template <int N>
__launch_bounds__(MAX_FFT_RESOLUTION/2)
__global__ void kernel_ComputeColumns  (float4* __restrict__ displacement_output)
{
	float4 displacement[2];
	computeColumns<N>(displacement);

	displacement_output += blockIdx.x + gConstants[N].m_resolution * threadIdx.x;
	displacement_output[0] = displacement[0];
	displacement_output[gConstants[N].m_half_of_resolution_squared] = displacement[1];
}

extern "C" 
hipError_t cuda_ComputeColumns(float4* displacement, int resolution, int constantsIndex, hipStream_t cu_stream)
{
	dim3 block = dim3(resolution/2);
    dim3 grid = dim3(resolution);
	int sharedMemory = 3 * sizeof(float) * resolution;

	switch(constantsIndex)
	{
		case 0: kernel_ComputeColumns<0><<<grid, block, sharedMemory, cu_stream>>>(displacement); break;
		case 1: kernel_ComputeColumns<1><<<grid, block, sharedMemory, cu_stream>>>(displacement); break;
		case 2: kernel_ComputeColumns<2><<<grid, block, sharedMemory, cu_stream>>>(displacement); break;
		case 3: kernel_ComputeColumns<3><<<grid, block, sharedMemory, cu_stream>>>(displacement); break;
	}
	return hipPeekAtLastError();
}

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 200

surface<void, cudaSurfaceType2D> gDisplacement;

template <int N>
__launch_bounds__(MAX_FFT_RESOLUTION/2)
__global__ void kernel_ComputeColumns_array()
{
	float4 displacement[2];
	computeColumns<N>(displacement);

	ushort4 displacement0 = make_ushort4(
		__float2half_rn(displacement[0].x),
		__float2half_rn(displacement[0].y),
		__float2half_rn(displacement[0].z),
		0);

	ushort4 displacement1 = make_ushort4(
		__float2half_rn(displacement[1].x),
		__float2half_rn(displacement[1].y),
		__float2half_rn(displacement[1].z),
		0);

	int rowAddr = blockIdx.x * sizeof(ushort4);
	surf2Dwrite(displacement0, gDisplacement, rowAddr, threadIdx.x);
	surf2Dwrite(displacement1, gDisplacement, rowAddr, threadIdx.x + gConstants[N].m_half_resolution);
}

extern "C" 
hipError_t cuda_ComputeColumns_array(hipArray* displacement, int resolution, int constantsIndex, hipStream_t cu_stream)
{
	cudaBindSurfaceToArray(gDisplacement, displacement);
    dim3 block = dim3(resolution/2);
    dim3 grid = dim3(resolution);
	int sharedMemory = 3 * sizeof(float) * resolution;

	switch(constantsIndex)
	{
		case 0: kernel_ComputeColumns_array<0><<<grid, block, sharedMemory, cu_stream>>>(); break;
		case 1: kernel_ComputeColumns_array<1><<<grid, block, sharedMemory, cu_stream>>>(); break;
		case 2: kernel_ComputeColumns_array<2><<<grid, block, sharedMemory, cu_stream>>>(); break;
		case 3: kernel_ComputeColumns_array<3><<<grid, block, sharedMemory, cu_stream>>>(); break;
	}
	return hipPeekAtLastError();
}

#endif